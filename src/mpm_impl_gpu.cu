#include "hip/hip_runtime.h"
#include "mpm_impl.h" // implement this header

#include "cu_hack.h"
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime_api.h"

/*
	// following 4 headers play NICE with NVCC+MSVC
	#include "Eigen/Core"
	#include "Eigen/LU"
	#include "Eigen/Cholesky"
	#include "Eigen/QR"
	
	// following 3 headers are NOT OK with NVCC+MSVC
	// Eigen/SVD is the VILLAIN
	#include "Eigen/SVD"
	#include "Eigen/Geometry"
	#include "Eigen/Eigenvalues"

	// so do not use <Eigen/Dense> directly
*/
#include <Eigen/core>

#include <cstdio>
#include <random>



__global__ void cu_initialize_p_kernel(float* x, float* v,
	float* C, float* F, int* material, float* Jp, float* color_device, unsigned int n_particles)
{
	const unsigned long long seed = 1;
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	// this condition is VITAL! 
	if (idx >= n_particles) return;

	int group_size = n_particles / 3;

	// this kernel will only be executed once at launch, care everything but performance
	hiprandState state;
	hiprand_init(seed, idx, 0, &state);
	float px, py;
	if (idx / group_size == 0) {
		px = 0.05f + 0 * 0.3f + hiprand_uniform(&state) * 0.999999f * 0.25f;
		py = 0.05f + 1 * 0.3f + hiprand_uniform(&state) * 0.999999f * 0.25f;
		material[idx] = 2;
		color_device[idx * 3 + 0] = 1.0f; // r
		color_device[idx * 3 + 1] = 0.976f; // g
		color_device[idx * 3 + 2] = 0.976f; // b
	}
	if (idx / group_size == 1) {
		px = 0.05f + 1 * 0.3f + hiprand_uniform(&state) * 0.999999f * 0.25f;
		py = 0.05f + 2 * 0.3f + hiprand_uniform(&state) * 0.999999f * 0.25f;
		material[idx] = 0;
		color_device[idx * 3 + 0] = 0.52f; // r
		color_device[idx * 3 + 1] = 0.80f; // g
		color_device[idx * 3 + 2] = 0.976f; // b
	}
	if (idx / group_size == 2) {
		px = 0.05f + 2 * 0.3f + hiprand_uniform(&state) * 0.999999f * 0.25f;
		py = 0.05f + 1 * 0.3f + hiprand_uniform(&state) * 0.999999f * 0.25f;
		material[idx] = 1;
		color_device[idx * 3 + 0] = 0.99f; // r
		color_device[idx * 3 + 1] = 0.7f; // g
		color_device[idx * 3 + 2] = 0.2f; // b
	}
	
	x[idx * 2 + 0] = px;
	x[idx * 2 + 1] = py;
	v[idx * 2 + 0] = v[idx * 2 + 1] = 0.0f;
	F[idx * 4 + 0] = F[idx * 4 + 3] = 1.0f;
	F[idx * 4 + 1] = F[idx * 4 + 2] = 0.0f;
	C[idx * 4 + 0] = C[idx * 4 + 1] = C[idx * 4 + 2] = C[idx * 4 + 3] = 0.0f;
	Jp[idx] = 1.0f; // fixed bug: if 0.0f here, only jelly will work. Wonder WHY?
	
}

__global__ void cu_initialize_grid_kernel(float* grid_v, float* grid_m, unsigned int n_grid)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	// this condition is VITAL! 
	if (idx >= n_grid * n_grid) return;

	grid_v[idx * 2 + 0] = 0.0f;
	grid_v[idx * 2 + 1] = 0.0f;
	grid_m[idx] = 0.0f;
}

void cu_initialize(float** x_device, float** v_device,
	float** C_device, float** F_device, int** material_device, float** Jp_device, float** color_device,
	unsigned int n_particles,
	float** grid_v_device, float** grid_m_device, unsigned int n_grid) 
{
	const unsigned int dim = 2;
	int n_threads = 128;

	// particles
	int n_blocks = (n_particles + n_threads - 1) / n_threads;
	if (*x_device == nullptr) {
		checkCudaErrors(hipMalloc(x_device, sizeof(float) * n_particles * dim));
	} // else we use GL_CUDA interop
	checkCudaErrors(hipMalloc(v_device, sizeof(float) * n_particles * dim));
	checkCudaErrors(hipMalloc(C_device, sizeof(float) * n_particles * dim * dim));
	checkCudaErrors(hipMalloc(F_device, sizeof(float) * n_particles * dim * dim));
	checkCudaErrors(hipMalloc(material_device, sizeof(int) * n_particles));
	checkCudaErrors(hipMalloc(Jp_device, sizeof(float) * n_particles));
	checkCudaErrors(hipMalloc(color_device, sizeof(float) * n_particles * 3));
	cu_initialize_p_kernel KERNEL_ARGS2(n_blocks, n_threads)
		(*x_device, *v_device, *C_device, *F_device, *material_device, *Jp_device, *color_device, n_particles);

	// grid
	n_blocks = (n_grid * n_grid + n_threads - 1) / n_threads;
	checkCudaErrors(hipMalloc(grid_v_device, sizeof(float) * n_grid * n_grid * dim));
	checkCudaErrors(hipMalloc(grid_m_device, sizeof(float) * n_grid * n_grid));
	cu_initialize_grid_kernel KERNEL_ARGS2(n_blocks, n_threads) (*grid_v_device, *grid_m_device, n_grid);
}




void cu_inigrid_substep(float* grid_v_device, float* grid_m_device, unsigned int n_grid) {
	const unsigned int dim = 2;
	int n_threads = 128;

	int n_blocks = (n_grid * n_grid + n_threads - 1) / n_threads;
	cu_initialize_grid_kernel KERNEL_ARGS2(n_blocks, n_threads) (grid_v_device, grid_m_device, n_grid);
	// checkCudaErrors(hipDeviceSynchronize());
}



__device__ void cu_svd22_raw(const float* a, float* u, float* s, float* v) {
	// const float a[4], float u[4], float s[2], float v[4]
	s[0] = (sqrtf(powf(a[0] - a[3], 2) + powf(a[1] + a[2], 2)) + 
		sqrtf(powf(a[0] + a[3], 2) + powf(a[1] - a[2], 2))) / 2.0f;
	s[1] = fabsf(s[0] - sqrtf(powf(a[0] - a[3], 2) + powf(a[1] + a[2], 2)));
	v[2] = (s[0] > s[1]) ? 
		sinf((atan2f(2 * (a[0] * a[1] + a[2] * a[3]), 
			a[0] * a[0] - a[1] * a[1] + a[2] * a[2] - a[3] * a[3])) / 2.0f) : 0.0f;
	v[0] = sqrtf(1 - v[2] * v[2]);
	v[1] = -v[2];
	v[3] = v[0];
	u[0] = (s[0] != 0) ? (a[0] * v[0] + a[1] * v[2]) / s[0] : 1.0f;
	u[2] = (s[0] != 0) ? (a[2] * v[0] + a[3] * v[2]) / s[0] : 0.0f;
	u[1] = (s[1] != 0) ? (a[0] * v[1] + a[1] * v[3]) / s[1] : -u[2];
	u[3] = (s[1] != 0) ? (a[2] * v[1] + a[3] * v[3]) / s[1] : u[0];
}




// be careful of using std::functions
__global__ void cu_p2g_substep_kernel(
	float* x, float* v, float* F, float* C, int* material, float* Jp, unsigned int n_particles,
	float* grid_v, float* grid_m, unsigned int n_grid,
	float dx, float inv_dx, float dt, float mu_0, float lambda_0, float p_vol, float p_mass )
{
	int p = blockDim.x * blockIdx.x + threadIdx.x;

	// this condition is VITAL! 
	if (p >= n_particles) return;

	Eigen::Vector2i base{ int(x[p * 2 + 0] * inv_dx - 0.5),
							  int(x[p * 2 + 1] * inv_dx - 0.5) };
	Eigen::Vector2f fx =
		Eigen::Vector2f(x[p * 2 + 0], x[p * 2 + 1]) * inv_dx - base.cast<float>();

	Eigen::Vector2f w[3]{
		0.5f * (1.5f - fx.array()) * (1.5f - fx.array()),
		0.75f - (fx.array() - 1.0f) * (fx.array() - 1.0f),
		0.5f * (fx.array() - 0.5f) * (fx.array() - 0.5f)
	};

	Eigen::Matrix2f temp_F;
	temp_F << F[p * 4 + 0], F[p * 4 + 1],
		F[p * 4 + 2], F[p * 4 + 3];

	Eigen::Matrix2f identity;
	identity.setIdentity();
	Eigen::Matrix2f temp_C;
	temp_C << C[p * 4 + 0], C[p * 4 + 1],
		C[p * 4 + 2], C[p * 4 + 3];

	temp_F = (identity + dt * temp_C) * (temp_F);

	float h;
	h = expf(10.0f * (1.0f - Jp[p]));
	if (material[p] == 1) {
		h = 0.3f;
	}

	float mu = mu_0 * h;
	float la = lambda_0 * h;
	if (material[p] == 0) {
		mu = 0.0f;
	}

	// SVD-related begin
	// we use analytical solution for 2x2 SVD instead of Eigen, test it on CPU first
	const float A[4]{ temp_F(0, 0), temp_F(0, 1) , temp_F(1, 0) , temp_F(1, 1) };
	float U_data[4], S_data[2], V_data[4];
	cu_svd22_raw(A, U_data, S_data, V_data);
	Eigen::Matrix2f U, V;
	U << U_data[0], U_data[1], U_data[2], U_data[3];
	V << V_data[0], V_data[1], V_data[2], V_data[3];

	float J = 1.0f;
	// shall we expand this for-loop?
	for (int i = 0; i < 2; i++) {
		float new_sig = S_data[i];
		if (material[p] == 2)
			new_sig = min(max(S_data[i], 1.0f - 2.5e-2f), 1.0f + 4.5e-3f);
		Jp[p] *= S_data[i] / new_sig;
		S_data[i] = new_sig;
		J *= new_sig;
	}
	// if (J < 0.01) { printf("%f", J); }
	
	if (material[p] == 0) {
		temp_F = identity * sqrtf(J);
	}
	else if (material[p] == 2) {
		Eigen::Matrix2f sig22;
		sig22 << S_data[0], 0, 0, S_data[1];
		temp_F = U * sig22 * V.transpose();
	}

	Eigen::Matrix2f stress, affine;
	stress = 2 * mu * (temp_F - U * V.transpose()) * temp_F.transpose() + identity * la * J * (J - 1);
	stress = (-dt * p_vol * 4 * inv_dx * inv_dx) * stress;
	affine = stress + p_mass * temp_C;
	// SVD-related end

	// shall we expand this for-loop?
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			Eigen::Vector2i offset;
			offset << i, j;
			Eigen::Vector2f dpos;
			dpos = (offset.cast<float>() - fx) * dx;
			float weight = w[i](0) * w[j](1);
			Eigen::Vector2i temp_index;
			temp_index = base + offset;
			Eigen::Vector2f temp_v, temp_gv;
			temp_v(0) = v[p * 2 + 0];
			temp_v(1) = v[p * 2 + 1];
			temp_gv = weight * (p_mass * temp_v + affine * dpos);
			unsigned int index = temp_index(0) * n_grid + temp_index(1);
			
			// atomic add!
			atomicAdd(&grid_v[index * 2 + 0], temp_gv(0));
			atomicAdd(&grid_v[index * 2 + 1], temp_gv(1));
			atomicAdd(&grid_m[index],  weight * p_mass);
			// grid_v[index * 2 + 0] += temp_gv(0);
			// grid_v[index * 2 + 1] += temp_gv(1);
			// grid_m[index] += weight * p_mass;
		}
	}

	F[p * 4 + 0] = temp_F(0, 0);
	F[p * 4 + 1] = temp_F(0, 1);
	F[p * 4 + 2] = temp_F(1, 0);
	F[p * 4 + 3] = temp_F(1, 1);
}


void cu_p2g_substep(
	float* x_device, float* v_device, float* F_device, float* C_device, int* material_device, float* Jp_device, unsigned int n_particles,
	float* grid_v_device, float* grid_m_device, unsigned int n_grid,
	float dx, float inv_dx, float dt, float mu_0, float lambda_0, float p_vol, float p_mass) {
	const unsigned int dim = 2;
	int n_threads = 128;

	// particles
	int n_blocks = (n_particles + n_threads - 1) / n_threads;

	// FIXME: error in fluid/snow material
	cu_p2g_substep_kernel KERNEL_ARGS2(n_blocks, n_threads)
		(x_device, v_device, F_device, C_device, material_device, Jp_device, n_particles, 
			grid_v_device, grid_m_device, n_grid, 
			dx, inv_dx, dt, mu_0, lambda_0, p_vol, p_mass);
	// checkCudaErrors(hipDeviceSynchronize());
}



__global__ void cu_boundary_substep_kernel(float* grid_v, float* grid_m, unsigned int n_grid,
	float dt, float gravity)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	// this condition is VITAL! 
	if (idx >= n_grid * n_grid) return;

	// idx = i * n_grid + j
	int i = idx / n_grid;
	int j = idx % n_grid;

	// the condition here might be a big overhead, we may optimize it
	
	if (grid_m[idx] > 0) {
		grid_v[idx * 2 + 0] = (1 / grid_m[idx]) * grid_v[idx * 2 + 0];
		grid_v[idx * 2 + 1] = (1 / grid_m[idx]) * grid_v[idx * 2 + 1];
		grid_v[idx * 2 + 1] -= dt * gravity * 1.0f;
		if (i < 3 && grid_v[idx * 2] < 0) {
			grid_v[idx * 2] = 0;
		}
		if (i > n_grid - 3 && grid_v[idx * 2] > 0) {
			grid_v[idx * 2] = 0;
		}
		if (j < 3 && grid_v[idx * 2 + 1] < 0) {
			grid_v[idx * 2 + 1] = 0;
		}
		if (j > n_grid - 3 && grid_v[idx * 2 + 1] > 0) {
			grid_v[idx * 2 + 1] = 0;
		}
	}
}

void cu_boundary_substep(float* grid_v_device, float* grid_m_device, unsigned int n_grid,
	float dt, float gravity)
{
	const unsigned int dim = 2;
	int n_threads = 128;

	int n_blocks = (n_grid * n_grid + n_threads - 1) / n_threads;
	cu_boundary_substep_kernel KERNEL_ARGS2(n_blocks, n_threads) 
		(grid_v_device, grid_m_device, n_grid, dt, gravity);
	// checkCudaErrors(hipDeviceSynchronize());
}

__global__ void cu_g2p_substep_kernel(
	float* x, float* v, float* C, unsigned int n_particles,
	float* grid_v, float* grid_m, unsigned int n_grid,
	float dt, float inv_dx)
{
	const unsigned int dim = 2;
	int p = blockDim.x * blockIdx.x + threadIdx.x;
	

	// this condition is VITAL! 
	if (p >= n_particles) return;

	Eigen::Vector2i base{ int(x[p * 2 + 0] * inv_dx - 0.5f),
							  int(x[p * 2 + 1] * inv_dx - 0.5f) };

	Eigen::Vector2f fx =
		Eigen::Vector2f(x[p * 2 + 0], x[p * 2 + 1]) * inv_dx - base.cast<float>();

	Eigen::Vector2f w[3] {
		0.5f * (1.5f - fx.array()) * (1.5f - fx.array()),
		0.75f - (fx.array() - 1.0f) * (fx.array() - 1.0f),
		0.5f * (fx.array() - 0.5f) * (fx.array() - 0.5f)
	};

	Eigen::Vector2f new_v; new_v.setZero();
	Eigen::Matrix2f new_c; new_c.setZero();
	// we may expand this for-loop
	// I got a feel that CUDA has little love with inner for-loop
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			Eigen::Vector2f dpos, vij;
			vij << float(i), float(j);
			dpos = vij - fx;
			unsigned int index = (base(0) + i) * n_grid + base(1) + j;
			
			Eigen::Vector2f g_v{ grid_v[index * 2 + 0], grid_v[index * 2 + 1] };
			float weight;
			weight = w[i](0) * w[j](1);
			new_v += weight * g_v;
			new_c += 4 * inv_dx * weight * g_v * dpos.transpose();
		}
	}
	v[p * 2 + 0] = new_v(0);
	v[p * 2 + 1] = new_v(1);
	C[p * 4 + 0] = new_c(0);
	C[p * 4 + 1] = new_c(1);
	C[p * 4 + 2] = new_c(2);
	C[p * 4 + 3] = new_c(3);
	x[p * 2 + 0] += dt * v[p * 2 + 0];
	x[p * 2 + 1] += dt * v[p * 2 + 1];
}

void cu_g2p_substep(
	float* x_device, float* v_device, float* C_device, unsigned int n_particles,
	float* grid_v_device, float* grid_m_device, unsigned int n_grid,
	float dt, float inv_dx) {
	const unsigned int dim = 2;
	int n_threads = 128;

	// particles
	int n_blocks = (n_particles + n_threads - 1) / n_threads;

	cu_g2p_substep_kernel KERNEL_ARGS2(n_blocks, n_threads)
		(x_device, v_device, C_device, n_particles, grid_v_device, grid_m_device, n_grid, dt, inv_dx);
	// checkCudaErrors(hipDeviceSynchronize());
}


// auxiliary functions

void arrcopy_gpu_to_cpu(
	float* x_device, float* v_device, float* C_device, float* F_device, int* material_device, float* Jp_device,
	float* x_host, float* v_host, float* C_host, float* F_host, int* material_host, float* Jp_host, unsigned int n_particles,
	float* grid_v_device, float* grid_m_device, float* grid_v_host, float* grid_m_host, unsigned int n_grid) {
	const unsigned int dim = 2;
	checkCudaErrors(hipMemcpy(x_host, x_device, sizeof(float) * n_particles * dim, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(v_host, v_device, sizeof(float) * n_particles * dim, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(C_host, C_device, sizeof(float) * n_particles * dim * dim, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(F_host, F_device, sizeof(float) * n_particles * dim * dim, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(material_host, material_device, sizeof(int) * n_particles, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(Jp_host, Jp_device, sizeof(float) * n_particles, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(grid_v_host, grid_v_device, sizeof(float) * n_grid * n_grid * dim, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(grid_m_host, grid_m_device, sizeof(float) * n_grid * n_grid, hipMemcpyDeviceToHost));
	
}

void arrcopy_cpu_to_gpu(
	float* x_device, float* v_device, float* C_device, float* F_device, int* material_device, float* Jp_device,
	float* x_host, float* v_host, float* C_host, float* F_host, int* material_host, float* Jp_host, unsigned int n_particles,
	float* grid_v_device, float* grid_m_device, float* grid_v_host, float* grid_m_host, unsigned int n_grid) {
	const unsigned int dim = 2;
	checkCudaErrors(hipMemcpy(x_device, x_host, sizeof(float) * n_particles * dim, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(v_device, v_host, sizeof(float) * n_particles * dim, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(C_device, C_host, sizeof(float) * n_particles * dim * dim, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(F_device, F_host, sizeof(float) * n_particles * dim * dim, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(material_device, material_host, sizeof(int) * n_particles, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(Jp_device, Jp_host, sizeof(float) * n_particles, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(grid_v_device, grid_v_host, sizeof(float) * n_grid * n_grid * dim, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(grid_m_device, grid_m_host, sizeof(float) * n_grid * n_grid, hipMemcpyHostToDevice));
}


void copy_x_from_gpu(float* x_device, float* x_host, unsigned int n_particles) {
	const unsigned int dim = 2;
	checkCudaErrors(hipMemcpy(x_host, x_device, sizeof(float) * n_particles * dim, hipMemcpyDeviceToHost));
}

void copy_color_from_gpu(float* color_device, float* color_host, unsigned int n_particles) {
	checkCudaErrors(hipMemcpy(color_host, color_device, sizeof(float) * n_particles * 3, hipMemcpyDeviceToHost));
}

void sync_all() {
	checkCudaErrors(hipDeviceSynchronize());
}