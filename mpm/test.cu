#include "hip/hip_runtime.h"
#include<glad/glad.h>
#include<GLFW/glfw3.h>
#include<glm/glm.hpp>
#include<glm/gtc/matrix_transform.hpp>
#include<glm/gtc/type_ptr.hpp>
#include"camera.h"
#include"shader_m.h"
#include<iostream>
#include<vector>
#include<map>
#include<hip/hip_runtime.h>
#include<>
#include"hip/hip_runtime_api.h"
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<Eigen/Core>
#include<cmath>
#include"svd3_cuda.h"
#include<cstdio>
#include<ticktock.h>
#include"render.h"
#include<fstream>


const int dim = 3;
const int n_particles = 50000;
const int n_grid = 64;
const int n_total_grid = n_grid * n_grid * n_grid;
int step = 80;
float dt = 1e-4;
float dx = 1.0f / n_grid;
float inv_dx = n_grid;
float p_rho = 1.0f;
float p_vol = (dx * 0.5f) * (dx * 0.5f);
float p_mass = p_vol * p_rho;
float gravity[] = { 0,-3.8,0 };
float E = 1000.0f;
float nu = 0.2f;
float mu_0 = E / (2.0f * (1.0f + nu));
float lambda_0 = E * nu / ((1.0f + nu) * (1.0f - 2.0f * nu));
int bound = 3;
const int WATER = 0;
const int JELLY = 1;
const int SNOW = 2;

const unsigned int sphere_row = 15;
const unsigned int sphere_col = 20;
float sphere_vertices[6 * sphere_row * sphere_col];  
unsigned int sphere_indices[6 * (sphere_row - 1) * sphere_col];

const unsigned int tool_row = 3;
const unsigned int tool_col = 20;
const int n_rigid_particles = 2 * (tool_row - 1) * tool_col;
const int n_tool_vertices = tool_row * tool_col;

//float tool_vertices_host_temp[6 * tool_row * tool_col];
float tool_vertices_host[6 * tool_row * tool_col];
unsigned int tool_indices_host[6 * (tool_row - 1) * tool_col];
float tool_center_host[dim] = { 0.6f, 0.68f, 0.60f };
float tool_center_init[dim] = { 0.6f, 0.68f, 0.60f };
float rigid_x_host[dim * n_rigid_particles];

float F_x_host[dim * n_particles];
float F_v_host[dim * n_particles];
float F_C_host[dim * dim * n_particles];
float F_dg_host[dim * dim * n_particles];
float F_Jp_host[n_particles];
float F_colors_host[3 * n_particles];
int F_materials_host[n_particles];
float F_grid_v_host[dim * n_total_grid];
float F_grid_m_host[n_total_grid];
int F_used_host[n_particles];

float C_grid_d_host[n_total_grid];
int C_grid_A_host[n_total_grid];
int C_grid_T_host[n_total_grid];
int C_grid_surface_host[n_total_grid];
float C_p_d_host[n_particles];
int C_p_A_host[n_particles];
int C_p_T_host[n_particles];
float C_p_n_host[dim * n_particles];

char* MCTable_path = "MC_Table.txt";
const int n_MC_grid = 64;
const int n_MC_total_grid = n_MC_grid * n_MC_grid * n_MC_grid;
float MC_dx = 1.0f / n_MC_grid;
float MC_inv_dx = n_MC_grid;
float MC_SDF_host[n_MC_total_grid];
int* n_MC_triangle_host;
int MC_et_host[256 * 4 * 3];
float MC_vertices_host[dim * 15 * n_MC_total_grid];
float MC_vertice_color_host[3 * 15 * n_MC_total_grid];
int MC_triangle[15 * n_MC_total_grid];
float MC_node_pos[dim * n_MC_total_grid];
float MC_node_color[3 * n_MC_total_grid];

unsigned int n_threads = 128;
unsigned int n_particle_blocks = (n_particles + n_threads - 1) / n_threads;
unsigned int n_grid_blocks = (n_total_grid + n_threads - 1) / n_threads;
unsigned int n_tool_vertices_blocks = (n_tool_vertices + n_threads - 1) / n_threads;
unsigned int n_rigid_blocks = (n_rigid_particles + n_threads - 1) / n_threads;
unsigned int n_MC_grid_blocks = (n_MC_total_grid + n_threads - 1) / n_threads;

void init_chuiti() {
    std::ifstream file;
    file.open("particles.ply");

    for (int i = 0; i < n_particles; i++) {
        file >> F_x_host[i * 3 + 0];
        F_x_host[i * 3 + 0] = F_x_host[i * 3 + 0] / 30.0 + 0.45;
        file >> F_x_host[i * 3 + 1];
        F_x_host[i * 3 + 1] = F_x_host[i * 3 + 1] / 30.0 - 4.65;
        file >> F_x_host[i * 3 + 2];
        F_x_host[i * 3 + 2] = F_x_host[i * 3 + 2] / 30.0;
    }
    file.close();
}

__global__ void init_particle(
    int n_particles,
    float* F_x_device,
    float* F_v_device,
    float* F_C_device,
    float* F_dg_device,
    float* F_Jp_device,
    float* F_colors_device,
    int* F_materials_device,
    int* F_used_device
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > n_particles)
        return;
    hiprandState state;
    hiprand_init(1, tid, 0, &state);
    int group_size = n_particles;
    //F_x_device[tid * 3 + 0] = hiprand_uniform(&state) * 0.3 + 0.3 + 0.1 * (tid / group_size);
    //F_x_device[tid * 3 + 1] = hiprand_uniform(&state) * 0.3 + 0.0 + 0.32 * (tid / group_size);
    //F_x_device[tid * 3 + 2] = hiprand_uniform(&state) * 0.3 + 0.3 + 0.1 * (tid / group_size);
    F_v_device[tid * 3 + 0] = 0.0f;
    F_v_device[tid * 3 + 1] = 0.0f;
    F_v_device[tid * 3 + 2] = 0.0f;
    F_C_device[tid * 9 + 0] = 0.0f;
    F_C_device[tid * 9 + 1] = 0.0f;
    F_C_device[tid * 9 + 2] = 0.0f;
    F_C_device[tid * 9 + 3] = 0.0f;
    F_C_device[tid * 9 + 4] = 0.0f;
    F_C_device[tid * 9 + 5] = 0.0f;
    F_C_device[tid * 9 + 6] = 0.0f;
    F_C_device[tid * 9 + 7] = 0.0f;
    F_C_device[tid * 9 + 8] = 0.0f;
    F_dg_device[tid * 9 + 0] = 1.0f;
    F_dg_device[tid * 9 + 1] = 0.0f;
    F_dg_device[tid * 9 + 2] = 0.0f;
    F_dg_device[tid * 9 + 3] = 0.0f;
    F_dg_device[tid * 9 + 4] = 1.0f;
    F_dg_device[tid * 9 + 5] = 0.0f;
    F_dg_device[tid * 9 + 6] = 0.0f;
    F_dg_device[tid * 9 + 7] = 0.0f;
    F_dg_device[tid * 9 + 8] = 1.0f;
    F_Jp_device[tid] = 1.0f;
    F_materials_device[tid] = 1;
    F_colors_device[tid * 3 + 0] = 0.93f;
    F_colors_device[tid * 3 + 1] = 0.33f;
    F_colors_device[tid * 3 + 2] = 0.13f;
    F_used_device[tid] = 1;
}

__global__ void init_rigid_particles(
    int n_rigid_particles,
    float* tool_vertices_device,
    unsigned int* tool_indices_device,
    float* tool_center_device,
    float* rigid_x_device
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > n_rigid_particles)
        return;
    int vid1 = tool_indices_device[tid * 3 + 0];
    int vid2 = tool_indices_device[tid * 3 + 1];
    int vid3 = tool_indices_device[tid * 3 + 2];
    rigid_x_device[tid * 3 + 0] = (tool_vertices_device[vid1 * 6 + 0] + tool_vertices_device[vid2 * 6 + 0] + tool_vertices_device[vid3 * 6 + 0]) / 3.0f;
    rigid_x_device[tid * 3 + 1] = (tool_vertices_device[vid1 * 6 + 1] + tool_vertices_device[vid2 * 6 + 1] + tool_vertices_device[vid3 * 6 + 1]) / 3.0f;
    rigid_x_device[tid * 3 + 2] = (tool_vertices_device[vid1 * 6 + 2] + tool_vertices_device[vid2 * 6 + 2] + tool_vertices_device[vid3 * 6 + 2]) / 3.0f;
}

__global__ void rigid_move(
    int n_rigid_particles,
    float* rigid_x_device,
    float dt
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > n_rigid_particles)
        return;
    rigid_x_device[tid * 3 + 1] -= dt;

}

__global__ void tool_vertices_move(
    int n_tool_vertices,
    float* tool_vertices_device,
    float dt
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > n_tool_vertices)
        return;
    tool_vertices_device[tid * 6 + 1] -= dt;
}

__device__ Eigen::Vector3f cross(Eigen::Vector3f a, Eigen::Vector3f b) {
    return Eigen::Vector3f(a.y() * b.z() - b.y() * a.z(), -(a.x() * b.z() - b.x() * a.z()), a.x() * b.y() - b.x() * a.y());
}

__global__ void grid_clear(
    int n_total_grid,
    float* F_grid_v_device,
    float* F_grid_m_device,
    float* C_grid_d_device,
    int* C_grid_A_device,
    int* C_grid_T_device,
    int* C_grid_surface_device
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > n_total_grid)
        return;
    F_grid_v_device[tid * 3 + 0] = 0.0f;
    F_grid_v_device[tid * 3 + 1] = 0.0f;
    F_grid_v_device[tid * 3 + 2] = 0.0f;
    F_grid_m_device[tid] = 0.0f;

    C_grid_d_device[tid] = 0.0f;
    C_grid_A_device[tid] = 0;
    C_grid_T_device[tid] = 0;
    C_grid_surface_device[tid] = -1;
}

__global__ void boundary_condition(
    int n_total_grid,
    float* F_grid_v_device,
    float* F_grid_m_device,
    float dt,
    int n_grid,
    int bound,
    float inv_dx
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > n_total_grid)
        return;
    if (F_grid_m_device[tid] > 0) {
        F_grid_v_device[tid * 3 + 0] /= F_grid_m_device[tid];
        F_grid_v_device[tid * 3 + 1] /= F_grid_m_device[tid];
        F_grid_v_device[tid * 3 + 2] /= F_grid_m_device[tid];
    }
    F_grid_v_device[tid * 3 + 1] -= dt * 3.8f;
    int g_x, g_y, g_z;
    g_x = tid / (n_grid * n_grid);
    g_y = (tid - g_x * n_grid * n_grid) / n_grid;
    g_z = tid % n_grid;
    if (g_x < bound && F_grid_v_device[tid * 3 + 0] < 0.0f) {
        F_grid_v_device[tid * 3 + 0] = 0.0f;
    }
    if (g_x > n_grid - bound && F_grid_v_device[tid * 3 + 0] > 0.0f) {
        F_grid_v_device[tid * 3 + 0] = 0.0f;
    }
    if (g_y < bound && F_grid_v_device[tid * 3 + 1] < 0.0f) {
        F_grid_v_device[tid * 3 + 1] = 0.0f;
    }
    if (g_y > n_grid - bound && F_grid_v_device[tid * 3 + 1] > 0.0f) {
        F_grid_v_device[tid * 3 + 1] = 0.0f;
    }
    if (g_z < bound && F_grid_v_device[tid * 3 + 2] < 0.0f) {
        F_grid_v_device[tid * 3 + 2] = 0.0f;
    }
    if (g_z > n_grid - bound && F_grid_v_device[tid * 3 + 2] > 0.0f) {
        F_grid_v_device[tid * 3 + 2] = 0.0f;
    }
    if (g_y > 0.5f * inv_dx && g_z < 0.49 * inv_dx) {
        F_grid_v_device[tid * 3 + 0] = 0.0f;
        F_grid_v_device[tid * 3 + 1] = 0.0f;
        F_grid_v_device[tid * 3 + 2] = 0.0f;
    }
}

__device__ float compute_particle_distance(
    Eigen::Vector3f particle_point,
    float* tool_center_device
) {
    Eigen::Vector3f center;
    center << tool_center_device[0], tool_center_device[1], tool_center_device[2];
    float distance = (particle_point - center).norm() - 0.08;
    return distance;
}
__device__ float compute_distance(
    Eigen::Vector3f point,
    Eigen::Vector3f proj_point
) {
    return abs((point - proj_point).norm());
}
__device__ bool compute_grid_T(
    Eigen::Vector3f plane_normal,
    Eigen::Vector3f point,
    Eigen::Vector3f proj_point
) {
    return (plane_normal.dot(point - proj_point) > 0);
}
__device__ bool is_valid(
    int surface,
    float* tool_vertices_device,
    unsigned int* tool_indices_device,
    Eigen::Vector3f proj_point
) {
    int index = 3 * surface;
    unsigned int idx_a = tool_indices_device[index + 0];
    unsigned int idx_b = tool_indices_device[index + 1];
    unsigned int idx_c = tool_indices_device[index + 2];
    Eigen::Vector3f a, b, c, ab, bc, ca, ap, bp, cp, temp1, temp2, temp3;
    a << tool_vertices_device[idx_a * 6 + 0],
        tool_vertices_device[idx_a * 6 + 1],
        tool_vertices_device[idx_a * 6 + 2];
    b << tool_vertices_device[idx_b * 6 + 0],
        tool_vertices_device[idx_b * 6 + 1],
        tool_vertices_device[idx_b * 6 + 2];
    c << tool_vertices_device[idx_c * 6 + 0],
        tool_vertices_device[idx_c * 6 + 1],
        tool_vertices_device[idx_c * 6 + 2];
    ab = b - a;
    bc = c - b;
    ca = a - c;
    ap = proj_point - a;
    bp = proj_point - b;
    cp = proj_point - c;
    temp1 = cross(ab, ap);
    temp2 = cross(bc, bp);
    temp3 = cross(ca, cp);
    return (temp1.dot(temp2) > 0 && temp2.dot(temp3) > 0);
}
__global__ void kernel(
    float* tool_vertices_device,
    unsigned int* tool_indices_device,
    float* rigid_x_device
) {
    int surface = 0;
    Eigen::Vector3f proj_point;
    int idx_a, idx_b, idx_c;
    idx_a = tool_indices_device[surface * 3 + 0];
    idx_b = tool_indices_device[surface * 3 + 1];
    idx_c = tool_indices_device[surface * 3 + 2];
    Eigen::Vector3f a, b, c, r;
    a << tool_vertices_device[idx_a * 6 + 0],
        tool_vertices_device[idx_a * 6 + 1],
        tool_vertices_device[idx_a * 6 + 2];
    b << tool_vertices_device[idx_b * 6 + 0],
        tool_vertices_device[idx_b * 6 + 1],
        tool_vertices_device[idx_b * 6 + 2];
    c << tool_vertices_device[idx_c * 6 + 0],
        tool_vertices_device[idx_c * 6 + 1],
        tool_vertices_device[idx_c * 6 + 2];
    r << rigid_x_device[surface * 3 + 0], rigid_x_device[surface * 3 + 1], rigid_x_device[surface * 3 + 2];
    printf("a: (%f,%f,%f)\n", a(0), a(1), a(2));
    printf("b: (%f,%f,%f)\n", b(0), b(1), b(2));
    printf("c: (%f,%f,%f)\n", c(0), c(1), c(2));
    printf("r: (%f,%f,%f)\n", r(0), r(1), r(2));
}

__device__ Eigen::Vector3f compute_proj_point(
    int surface,
    float* tool_vertices_device,
    unsigned int* tool_indices_device,
    Eigen::Vector3f plane_normal,
    Eigen::Vector3f point
) {
    int index = tool_indices_device[3 * surface];
    Eigen::Vector3f plane_point;
    plane_point <<
        tool_vertices_device[index * 6 + 0],
        tool_vertices_device[index * 6 + 1],
        tool_vertices_device[index * 6 + 2];
    float A = plane_normal(0);
    float B = plane_normal(1);
    float C = plane_normal(2);
    float D = - A * plane_point(0) - B * plane_point(1) - C * plane_point(2);
    float temp = (A * A + B * B + C * C);
    float proj_x = ((B * B + C * C) * point(0) - A * (B * point(1) + C * point(2) + D)) / temp;
    float proj_y = ((A * A + C * C) * point(1) - B * (A * point(0) + C * point(2) + D)) / temp;
    float proj_z = ((A * A + B * B) * point(2) - C * (A * point(0) + B * point(1) + D)) / temp;
    Eigen::Vector3f proj_point;
    proj_point << proj_x, proj_y, proj_z;
    return proj_point;
}

__device__ Eigen::Vector3f compute_plane_normal(
    int surface,
    float* tool_vertices_device,
    unsigned int* tool_indices_device
) {
    int index = 3 * surface;
    unsigned int idx_a = tool_indices_device[index + 0];
    unsigned int idx_b = tool_indices_device[index + 1];
    unsigned int idx_c = tool_indices_device[index + 2];
    Eigen::Vector3f ab, bc, re;
    ab <<
        tool_vertices_device[idx_b * 6 + 0] - tool_vertices_device[idx_a * 6 + 0],
        tool_vertices_device[idx_b * 6 + 1] - tool_vertices_device[idx_a * 6 + 1],
        tool_vertices_device[idx_b * 6 + 2] - tool_vertices_device[idx_a * 6 + 2];
    bc <<
        tool_vertices_device[idx_c * 6 + 0] - tool_vertices_device[idx_b * 6 + 0],
        tool_vertices_device[idx_c * 6 + 1] - tool_vertices_device[idx_b * 6 + 1],
        tool_vertices_device[idx_c * 6 + 2] - tool_vertices_device[idx_b * 6 + 2];
    re = cross(ab, bc);
    return re.normalized();
}

__global__ void grid_CDF(
    int n_rigid_particles,
    float* tool_vertices_device,
    unsigned int* tool_indices_device,
    float* tool_center_device,
    float* rigid_x_device,
    float dx,
    float inv_dx,
    float* C_grid_d_device,
    int* C_grid_A_device,
    int* C_grid_T_device,
    int* C_grid_surface_device,
    int n_grid
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > n_rigid_particles)
        return;
    Eigen::Vector3f rp;
    rp << rigid_x_device[tid * 3 + 0], rigid_x_device[tid * 3 + 1], rigid_x_device[tid * 3 + 2];
    Eigen::Vector3i base = (rp.array() * inv_dx - 0.5f).cast<int>();
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            for (int k = 0; k < 3; k++) {
                Eigen::Vector3i offset;
                offset << i, j, k;
                Eigen::Vector3f grid_node;
                grid_node = (offset + base).cast<float>() * dx;
                Eigen::Vector3f plane_normal, proj_point;
                plane_normal = compute_plane_normal(tid, tool_vertices_device, tool_indices_device);
                proj_point = compute_proj_point(
                    tid,
                    tool_vertices_device,
                    tool_indices_device,
                    plane_normal,
                    grid_node
                );
                if (is_valid(tid, tool_vertices_device, tool_indices_device, proj_point) == true) {
                int temp_index = (base + offset)(0) * n_grid * n_grid + (base + offset)(1) * n_grid + (base + offset)(2);
                    C_grid_A_device[temp_index] = 1;
                    float distance = compute_distance(grid_node, proj_point);
                    if (C_grid_surface_device[temp_index] == -1 || C_grid_d_device[temp_index] > distance) {
                        C_grid_d_device[temp_index] = distance;
                        C_grid_surface_device[temp_index] = tid;
                        if (compute_grid_T(plane_normal, grid_node, proj_point)) {
                            C_grid_T_device[temp_index] = 1;
                        }
                        else {
                            C_grid_T_device[temp_index] = -1;
                        }
                    }
                }
            }
        }
    }
}



__global__ void p2g(
    float* F_x_device,
    float* F_v_device,
    float* F_C_device,
    float* F_dg_device,
    float* F_Jp_device,
    float* F_colors_device,
    int* F_materials_device,
    float* F_grid_v_device,
    float* F_grid_m_device,
    int* F_used_device,
    float dt,
    int n_particles,
    float dx,
    float p_rho,
    float p_vol,
    float p_mass,
    float nu,
    float mu_0,
    float lambda_0,
    int WATER,
    int JELLY,
    int SNOW,
    float* C_grid_d_device,
    int* C_grid_A_device,
    int* C_grid_T_device,
    int* C_grid_surface_device,
    float* C_p_d_device,
    int* C_p_A_device,
    int* C_p_T_device,
    float* C_p_n_device,
    float* tool_center_device,
    int n_grid
) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i > n_particles) {
        return;
    }
    F_colors_device[i * 3 + 0] = 0.93f;
    F_colors_device[i * 3 + 1] = 0.33f;
    F_colors_device[i * 3 + 2] = 0.13f;
    Eigen::Vector3f Xp;
    Eigen::Vector3f F_xp(F_x_device[i * 3 + 0], F_x_device[i * 3 + 1], F_x_device[i * 3 + 2]);
    Xp = F_xp / dx;
    Eigen::Vector3i base = (Xp.array() - 0.5f).cast<int>();
    Eigen::Vector3f fx = Xp - base.cast<float>();
    Eigen::Vector3f w[3]{
        0.5f * (1.5f - fx.array()) * (1.5f - fx.array()),
        0.75f - (fx.array() - 1.0f) * (fx.array() - 1.0f),
        0.5f * (fx.array() - 0.5f) * (fx.array() - 0.5f)
    };

    //CDF=============
    C_p_A_device[i] = 0;
    C_p_T_device[i] = 0;
    C_p_d_device[i] = 0.0f;
    float Tpr = 0.0f;
    for (int ii = 0; ii < 3; ii++) {
        for (int jj = 0; jj < 3; jj++) {
            for (int kk = 0; kk < 3; kk++) {
                Eigen::Vector3i offset;
                offset << ii, jj, kk;
                int temp_index = (base + offset)(0) * n_grid * n_grid + (base + offset)(1) * n_grid + (base + offset)(2);
                if (C_grid_A_device[temp_index] == 1) {
                    C_p_A_device[i] = 1;
                }
                float weight = w[ii](0) * w[jj](1) * w[kk](2);
                Tpr += weight * C_grid_d_device[temp_index] * C_grid_T_device[temp_index];
            }
        }
    }
    C_p_d_device[i] = abs(Tpr);
    if (C_p_A_device[i] == 1) {
        C_p_d_device[i] = compute_particle_distance(Xp, tool_center_device);
        if (Tpr > 0) {
            C_p_T_device[i] = 1;
            F_colors_device[i * 3 + 1] = 1.0f;
        }
        else {
            C_p_T_device[i] = -1;
            F_colors_device[i * 3 + 2] = 1.0f;
        }
    }
    //=============


    Eigen::Matrix3f F_dgp;
    F_dgp <<
        F_dg_device[i * 9 + 0], F_dg_device[i * 9 + 1], F_dg_device[i * 9 + 2],
        F_dg_device[i * 9 + 3], F_dg_device[i * 9 + 4], F_dg_device[i * 9 + 5],
        F_dg_device[i * 9 + 6], F_dg_device[i * 9 + 7], F_dg_device[i * 9 + 8];
    Eigen::Matrix3f F_Cp;
    F_Cp <<
        F_C_device[i * 9 + 0], F_C_device[i * 9 + 1], F_C_device[i * 9 + 2],
        F_C_device[i * 9 + 3], F_C_device[i * 9 + 4], F_C_device[i * 9 + 5],
        F_C_device[i * 9 + 6], F_C_device[i * 9 + 7], F_C_device[i * 9 + 8];
    F_dgp = (Eigen::Matrix3f::Identity() + dt * F_Cp) * F_dgp;
    float h = expf(10.0f * (1.0f - F_Jp_device[i]));
    if (F_materials_device[i] == JELLY) {
        h = 0.3f;
    }
    float mu = mu_0 * h;
    float la = lambda_0 * h;
    if (F_materials_device[i] == WATER) {
        mu = 0.0f;
    }

    /*todo: SVD*/
    Eigen::Matrix3f U, sig, V;
    float u11, u12, u13,
        u21, u22, u23,
        u31, u32, u33;
    float s11, s22, s33;
    float v11, v12, v13,
        v21, v22, v23,
        v31, v32, v33;
    svd_cu(F_dgp(0, 0), F_dgp(0, 1), F_dgp(0, 2),
        F_dgp(1, 0), F_dgp(1, 1), F_dgp(1, 2),
        F_dgp(2, 0), F_dgp(2, 1), F_dgp(2, 2),
        u11, u12, u13,
        u21, u22, u23,
        u31, u32, u33,
        s11, s22, s33,
        v11, v12, v13,
        v21, v22, v23,
        v31, v32, v33);
    U << u11, u12, u13,
        u21, u22, u23,
        u31, u32, u33;
    sig << s11, 0, 0,
        0, s22, 0,
        0, 0, s33;
    V << v11, v12, v13,
        v21, v22, v23,
        v31, v32, v33;

    float J = 1.0f;
    for (int d = 0; d < 3; d++) {
        float new_sig = sig(d, d);
        if (F_materials_device[i] == SNOW) {
            new_sig = min(max(sig(d, d), 1.0f - 2.5e-2f), 1.0f + 4.5e-3f);
        }
        F_Jp_device[i] *= sig(d, d) / new_sig;
        sig(d, d) = new_sig;
        J *= new_sig;
    }
    if (F_materials_device[i] == WATER) {
        Eigen::Matrix3f new_F = Eigen::Matrix3f::Identity();
        new_F(0, 0) = J;
        F_dgp = new_F;
    }
    else if (F_materials_device[i] == SNOW) {
        F_dgp = U * sig * V.transpose();
    }
    Eigen::Matrix3f stress;
    stress = 2 * mu * (F_dgp - U * V.transpose()) * F_dgp.transpose()
        + Eigen::Matrix3f::Identity() * la * J * (J - 1);
    stress = (-dt * p_vol * 4) * stress / (dx * dx);
    Eigen::Matrix3f affine;
    affine = stress + p_mass * F_Cp;

    for (int ii = 0; ii < 3; ii++) {
        for (int jj = 0; jj < 3; jj++) {
            for (int kk = 0; kk < 3; kk++) {
                Eigen::Vector3f dpos;
                Eigen::Vector3i offset;
                offset << ii, jj, kk;

                int temp_index = (base + offset)(0) * n_grid * n_grid + (base + offset)(1) * n_grid + (base + offset)(2);
                if (C_p_T_device[i] * C_grid_T_device[temp_index] == -1) {
                    continue;
                }

                dpos = (offset.cast<float>() - fx) * dx;
                float weight = w[ii](0) * w[jj](1) * w[kk](2);
                Eigen::Vector3f F_grid_v, F_vp;
                F_vp << F_v_device[i * 3 + 0], F_v_device[i * 3 + 1], F_v_device[i * 3 + 2];
                F_grid_v = weight * (p_mass * F_vp + affine * dpos);
                atomicAdd(&F_grid_v_device[temp_index * 3 + 0], F_grid_v(0));
                atomicAdd(&F_grid_v_device[temp_index * 3 + 1], F_grid_v(1));
                atomicAdd(&F_grid_v_device[temp_index * 3 + 2], F_grid_v(2));
                atomicAdd(&F_grid_m_device[temp_index], weight * p_mass);
            }
        }
    }
    F_dg_device[i * 9 + 0] = F_dgp(0, 0);
    F_dg_device[i * 9 + 1] = F_dgp(0, 1);
    F_dg_device[i * 9 + 2] = F_dgp(0, 2);
    F_dg_device[i * 9 + 3] = F_dgp(1, 0);
    F_dg_device[i * 9 + 4] = F_dgp(1, 1);
    F_dg_device[i * 9 + 5] = F_dgp(1, 2);
    F_dg_device[i * 9 + 6] = F_dgp(2, 0);
    F_dg_device[i * 9 + 7] = F_dgp(2, 1);
    F_dg_device[i * 9 + 8] = F_dgp(2, 2);
}
__global__ void g2p(
    float* F_x_device,
    float* F_C_device,
    float* F_v_device,
    float* F_colors_device,
    int* F_materials_device,
    int* F_used_device,
    float* F_grid_v_device,
    float* F_grid_m_device,
    int n_particles,
    int n_grid,
    float dx,
    float dt,
    float* C_grid_d_device,
    int* C_grid_A_device,
    int* C_grid_T_device,
    int* C_grid_surface_device,
    float* C_p_d_device,
    int* C_p_A_device,
    int* C_p_T_device,
    float* C_p_n_device,
    float* tool_center_device,
    float p_mass
) {
    int p = blockDim.x * blockIdx.x + threadIdx.x;
    if (p > n_particles) 
        return;

    Eigen::Vector3f Xp;
    Eigen::Vector3f F_xp(F_x_device[p * 3 + 0], F_x_device[p * 3 + 1], F_x_device[p * 3 + 2]);
    Xp = F_xp / dx;
    Eigen::Vector3i base = (Xp.array() - 0.5f).cast<int>();
    Eigen::Vector3f fx = Xp - base.cast<float>();
    Eigen::Vector3f w[3]{
        0.5f * (1.5f - fx.array()) * (1.5f - fx.array()),
        0.75f - (fx.array() - 1.0f) * (fx.array() - 1.0f),
        0.5f * (fx.array() - 0.5f) * (fx.array() - 0.5f)
    };

    Eigen::Vector3f new_v = Eigen::Vector3f::Zero();
    Eigen::Matrix3f new_C = Eigen::Matrix3f::Zero();

    Eigen::Vector3f cp, tool_center, np;
    tool_center << tool_center_device[0], tool_center_device[1], tool_center_device[2];
    cp = Xp - tool_center;
    np = cp.normalized() * C_p_T_device[p];

    Eigen::Vector3f vp;
    vp << F_v_device[p * 3 + 0], F_v_device[p * 3 + 1], F_v_device[p * 3 + 2];
    for (int ii = 0; ii < 3; ii++) {
        for (int jj = 0; jj < 3; jj++) {
            for (int kk = 0; kk < 3; kk++) {
                Eigen::Vector3f dpos;
                Eigen::Vector3i offset;
                offset << ii, jj, kk;
                dpos = (offset.cast<float>() - fx) * dx;
                float weight = 1.0f;
                weight = w[ii](0) * w[jj](1) * w[kk](2);
                int temp_index = (base + offset)(0) * n_grid * n_grid + (base + offset)(1) * n_grid + (base + offset)(2);
                Eigen::Vector3f g_v;
                g_v.setZero();

                if (C_p_T_device[p] * C_grid_T_device[temp_index] == -1) {
                    float sg;
                    sg = vp.dot(np);
                    if (sg > 0) {
                        g_v = vp;
                    }
                    else {
                        g_v = vp - vp.dot(np) * np;
                    }
                    if (C_p_T_device[p] * C_p_d_device[p] > 0) {
                        g_v += np * 5;
                    }
                }
                else {
                    g_v << F_grid_v_device[temp_index * 3 + 0], F_grid_v_device[temp_index * 3 + 1], F_grid_v_device[temp_index * 3 + 2];
                }
                //g_v << F_grid_v_device[temp_index * 3 + 0], F_grid_v_device[temp_index * 3 + 1], F_grid_v_device[temp_index * 3 + 2];
                new_v += weight * g_v;
                new_C += 4 * weight * g_v * dpos.transpose() / (dx * dx);
            }
        }
    }
    F_v_device[p * 3 + 0] = new_v(0);
    F_v_device[p * 3 + 1] = new_v(1);
    F_v_device[p * 3 + 2] = new_v(2);
    if (C_p_A_device[p] == 1 && C_p_T_device[p] * C_p_d_device[p] < 0) {
        float kh = 0.001f;
        Eigen::Vector3f f_penalty;
        f_penalty = -kh * np * C_p_d_device[p] * C_p_T_device[p];
        F_v_device[p * 3 + 0] += dt * f_penalty(0) / p_mass;
        F_v_device[p * 3 + 1] += dt * f_penalty(1) / p_mass;
        F_v_device[p * 3 + 2] += dt * f_penalty(2) / p_mass;

    }
    F_x_device[p * 3 + 0] += dt * F_v_device[p * 3 + 0];
    F_x_device[p * 3 + 1] += dt * F_v_device[p * 3 + 1];
    F_x_device[p * 3 + 2] += dt * F_v_device[p * 3 + 2];
    F_C_device[p * 9 + 0] = new_C(0, 0);
    F_C_device[p * 9 + 1] = new_C(0, 1);
    F_C_device[p * 9 + 2] = new_C(0, 2);
    F_C_device[p * 9 + 3] = new_C(1, 0);
    F_C_device[p * 9 + 4] = new_C(1, 1);
    F_C_device[p * 9 + 5] = new_C(1, 2);
    F_C_device[p * 9 + 6] = new_C(2, 0);
    F_C_device[p * 9 + 7] = new_C(2, 1);
    F_C_device[p * 9 + 8] = new_C(2, 2);
}




void read_MCTable() {
    std::ifstream file;
    file.open(MCTable_path);
    for (int i = 0; i < 256; i++) {
        int index = i * 12;
        for (int j = 0; j < 12; j++) {
            file >> MC_et_host[index + j];
        }
        float temp;
        file >> temp;
    }
    file.close();
}

__global__ void compute_implicit_face(
    int n_MC_grid,
    int n_MC_total_grid,
    int n_particles,
    float MC_dx,
    float* F_x_device,
    float* MC_SDF_device
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > n_MC_total_grid)
        return;
    float min_dis = 10.0f;
    Eigen::Vector3f node_pos;
    int g_x = tid / (n_MC_grid * n_MC_grid);
    int g_y = (tid - g_x * n_MC_grid * n_MC_grid) / n_MC_grid;
    int g_z = tid % n_MC_grid;
    node_pos << g_x * MC_dx, g_y* MC_dx, g_z* MC_dx;
    for (int p = 0; p < n_particles; p++) {
        Eigen::Vector3f xp;
        xp << F_x_device[p * 3 + 0], F_x_device[p * 3 + 1], F_x_device[p * 3 + 2];
        float distance = (xp - node_pos).norm() - 0.02;
        if(distance < min_dis){
            min_dis = distance;
        }
    }
    MC_SDF_device[tid] = min_dis;
}

__device__ Eigen::Vector3f compute_MC_vertice(
    int tid,
    int n_MC_grid,
    float MC_dx,
    float* MC_SDF_device,
    int edge
) {
    int g_x = tid / (n_MC_grid * n_MC_grid);
    int g_y = (tid - g_x * n_MC_grid * n_MC_grid) / n_MC_grid;
    int g_z = tid % n_MC_grid;
    float a = abs(MC_SDF_device[tid]);
    float b = abs(MC_SDF_device[tid + n_MC_grid * n_MC_grid]);
    float c = abs(MC_SDF_device[tid + n_MC_grid * n_MC_grid + 1]);
    float d = abs(MC_SDF_device[tid + 1]);
    float e = abs(MC_SDF_device[tid + n_MC_grid]);
    float f = abs(MC_SDF_device[tid + n_MC_grid * n_MC_grid + n_MC_grid]);
    float g = abs(MC_SDF_device[tid + n_MC_grid * n_MC_grid + n_MC_grid + 1]);
    float h = abs(MC_SDF_device[tid + n_MC_grid + 1]);
    Eigen::Vector3f res_pos, base;
    base << g_x * MC_dx, g_y* MC_dx, g_z* MC_dx;
    if (edge == 0) {
        float temp = a / (a + b);
        res_pos = base + Eigen::Vector3f(MC_dx * temp, 0, 0);
    }
    if (edge == 1) {
        float temp = b / (b + c);
        res_pos = base + Eigen::Vector3f(MC_dx, 0, MC_dx * temp);
    }
    if (edge == 2) {
        float temp = d / (c + d);
        res_pos = base + Eigen::Vector3f(MC_dx * temp, 0, MC_dx);
    }
    if (edge == 3) {
        float temp = a / (a + d);
        res_pos = base + Eigen::Vector3f(0, 0, MC_dx * temp);
    }
    if (edge == 4) {
        float temp = e / (e + f);
        res_pos = base + Eigen::Vector3f(MC_dx * temp, MC_dx, 0);
    }
    if (edge == 5) {
        float temp = f / (f + g);
        res_pos = base + Eigen::Vector3f(MC_dx, MC_dx, MC_dx * temp);
    }
    if (edge == 6) {
        float temp = h / (h + g);
        res_pos = base + Eigen::Vector3f(MC_dx * temp, MC_dx, MC_dx);
    }
    if (edge == 7) {
        float temp = e / (e + h);
        res_pos = base + Eigen::Vector3f(0, MC_dx, MC_dx * temp);
    }
    if (edge == 8) {
        float temp = a / (a + e);
        res_pos = base + Eigen::Vector3f(0, MC_dx * temp, 0);
    }
    if (edge == 9) {
        float temp = b / (b + f);
        res_pos = base + Eigen::Vector3f(MC_dx, MC_dx * temp, 0);
    }
    if (edge == 10) {
        float temp = c / (c + g);
        res_pos = base + Eigen::Vector3f(MC_dx, MC_dx * temp, MC_dx);
    }
    if (edge == 11) {
        float temp = d / (d + h);
        res_pos = base + Eigen::Vector3f(0, MC_dx * temp, MC_dx);
    }
    return res_pos;
}


__global__ void implicit_to_explicit(
    int n_MC_grid,
    int n_MC_total_grid,
    float* MC_SDF_device,
    int* n_MC_triangle_device,
    int* MC_et_device,
    float* MC_vertices_device,
    float MC_dx
) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > n_MC_total_grid)
        return;
    int g_x = tid / (n_MC_grid * n_MC_grid);
    int g_y = (tid - g_x * n_MC_grid * n_MC_grid) / n_MC_grid;
    int g_z = tid % n_MC_grid;
    if (g_x == n_MC_grid - 1 || g_y == n_MC_grid - 1 || g_z == n_MC_grid - 1)
        return;
    int id = 0;
    if (MC_SDF_device[tid] > 0)
        id |= 1;
    if (MC_SDF_device[tid + n_MC_grid * n_MC_grid] > 0)
        id |= 2;
    if (MC_SDF_device[tid + n_MC_grid * n_MC_grid + 1] > 0)
        id |= 4;
    if (MC_SDF_device[tid + 1] > 0)
        id |= 8;
    if (MC_SDF_device[tid + n_MC_grid] > 0)
        id |= 16;
    if (MC_SDF_device[tid + n_MC_grid * n_MC_grid + n_MC_grid] > 0)
        id |= 32;
    if (MC_SDF_device[tid + n_MC_grid * n_MC_grid + n_MC_grid + 1] > 0)
        id |= 64;
    if (MC_SDF_device[tid + n_MC_grid + 1] > 0)
        id |= 128;
    for (int i = 0; i < 4; i++) {
        int temp = id * 4 + i;
        if (MC_et_device[temp * 3 + 0] > 0) {
            int index = atomicAdd(n_MC_triangle_device, 1);
            Eigen::Vector3f t1, t2, t3, normal;
            t1 = compute_MC_vertice(tid, n_MC_grid, MC_dx, MC_SDF_device, MC_et_device[temp * 3 + 0]);
            t2 = compute_MC_vertice(tid, n_MC_grid, MC_dx, MC_SDF_device, MC_et_device[temp * 3 + 1]);
            t3 = compute_MC_vertice(tid, n_MC_grid, MC_dx, MC_SDF_device, MC_et_device[temp * 3 + 2]);
            normal = cross(t3 - t1, t2 - t1).normalized();
            MC_vertices_device[index * 18 + 0] = t1(0);
            MC_vertices_device[index * 18 + 1] = t1(1);
            MC_vertices_device[index * 18 + 2] = t1(2);
            MC_vertices_device[index * 18 + 3] = normal(0);
            MC_vertices_device[index * 18 + 4] = normal(1);
            MC_vertices_device[index * 18 + 5] = normal(2);

            MC_vertices_device[index * 18 + 6] = t2(0);
            MC_vertices_device[index * 18 + 7] = t2(1);
            MC_vertices_device[index * 18 + 8] = t2(2);
            MC_vertices_device[index * 18 + 9] = normal(0);
            MC_vertices_device[index * 18 + 10] = normal(1);
            MC_vertices_device[index * 18 + 11] = normal(2);


            MC_vertices_device[index * 18 + 12] = t3(0);
            MC_vertices_device[index * 18 + 13] = t3(1);
            MC_vertices_device[index * 18 + 14] = t3(2);
            MC_vertices_device[index * 18 + 15] = normal(0);
            MC_vertices_device[index * 18 + 16] = normal(1);
            MC_vertices_device[index * 18 + 17] = normal(2);
            //printf("%f\n", t1(0));
        }

        //int temp = 12 * id + i * 3;

    }

}


std::map<std::vector<float>, int> vertice_map;
unsigned int MC_indices[n_MC_total_grid * 4 * 3] = { 0 };
void smooth_normal() {
    memset(MC_indices, 0, sizeof(MC_indices));
    for (int i = 0; i < *n_MC_triangle_host; i++) {
        for (int j = 0; j < 3; j++) {
            int index = i * 18 + j * 6;
            std::vector<float> vPos(3);
            vPos[0] = MC_vertices_host[index + 0];
            vPos[1] = MC_vertices_host[index + 1];
            vPos[2] = MC_vertices_host[index + 2];
            if (vertice_map.count(vPos) == 0) {
                int value = i * 3 + j;
                vertice_map[vPos] = value;
                MC_indices[value] = value;
            }
            else {
                int value = i * 3 + j;
                MC_indices[value] = vertice_map[vPos];
                MC_vertices_host[vertice_map[vPos] * 6 + 3] += MC_vertices_host[index + 3];
                MC_vertices_host[vertice_map[vPos] * 6 + 4] += MC_vertices_host[index + 4];
                MC_vertices_host[vertice_map[vPos] * 6 + 5] += MC_vertices_host[index + 5];
            }
        }
    }
}

int main() {
    read_MCTable();

    float* F_x_device;
    float* F_v_device;
    float* F_C_device;
    float* F_dg_device;
    float* F_Jp_device;
    float* F_colors_device;
    int* F_materials_device;
    float* F_grid_v_device;
    float* F_grid_m_device;
    int* F_used_device;

    float* C_grid_d_device;
    int* C_grid_A_device;
    int* C_grid_T_device;
    int* C_grid_surface_device;
    float* C_p_d_device;
    int* C_p_A_device;
    int* C_p_T_device;
    float* C_p_n_device;
    
    float* tool_vertices_device;
    unsigned int* tool_indices_device;
    float* tool_center_device;
    float* rigid_x_device;

    float* MC_SDF_device;
    int* n_MC_triangle_device;
    int* MC_et_device;
    float* MC_vertices_device;

    checkCudaErrors(hipMalloc(&F_x_device, dim * n_particles * sizeof(float)));
    checkCudaErrors(hipMalloc(&F_v_device, dim * n_particles * sizeof(float)));
    checkCudaErrors(hipMalloc(&F_C_device, dim * dim * n_particles * sizeof(float)));
    checkCudaErrors(hipMalloc(&F_dg_device, dim * dim * n_particles * sizeof(float)));
    checkCudaErrors(hipMalloc(&F_Jp_device, n_particles * sizeof(float)));
    checkCudaErrors(hipMalloc(&F_colors_device, 3 * n_particles * sizeof(float)));
    checkCudaErrors(hipMalloc(&F_materials_device, n_particles * sizeof(int)));
    checkCudaErrors(hipMalloc(&F_grid_v_device, dim * n_total_grid * sizeof(float)));
    checkCudaErrors(hipMalloc(&F_grid_m_device, n_total_grid * sizeof(float)));
    checkCudaErrors(hipMalloc(&F_used_device, n_particles * sizeof(int)));

    checkCudaErrors(hipMalloc(&C_grid_d_device, n_total_grid * sizeof(float)));
    checkCudaErrors(hipMalloc(&C_grid_A_device, n_total_grid * sizeof(int)));
    checkCudaErrors(hipMalloc(&C_grid_T_device, n_total_grid * sizeof(int)));
    checkCudaErrors(hipMalloc(&C_grid_surface_device, n_total_grid * sizeof(int)));
    checkCudaErrors(hipMalloc(&C_p_d_device, n_particles * sizeof(float)));
    checkCudaErrors(hipMalloc(&C_p_A_device, n_particles * sizeof(int)));
    checkCudaErrors(hipMalloc(&C_p_T_device, n_particles * sizeof(int)));
    checkCudaErrors(hipMalloc(&C_p_n_device, dim * n_particles * sizeof(float)));

    checkCudaErrors(hipMalloc(&tool_vertices_device, 6 * tool_row * tool_col * sizeof(float)));
    checkCudaErrors(hipMalloc(&tool_indices_device, 6 * (tool_row - 1) * tool_col * sizeof(unsigned int)));
    checkCudaErrors(hipMalloc(&tool_center_device, dim * sizeof(float)));
    checkCudaErrors(hipMalloc(&rigid_x_device, dim * n_rigid_particles * sizeof(float)));

    checkCudaErrors(hipMalloc(&MC_SDF_device, n_MC_total_grid * sizeof(float)));
    checkCudaErrors(hipMalloc(&n_MC_triangle_device, sizeof(int)));
    checkCudaErrors(hipMalloc(&MC_et_device, 256 * 12 * sizeof(int)));
    checkCudaErrors(hipMalloc(&MC_vertices_device, dim * n_MC_total_grid * 15 * sizeof(float)));
    
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glfwSetCursorPosCallback(window, mouse_callback);
    glfwSetScrollCallback(window, scroll_callback);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }
    glEnable(GL_DEPTH_TEST);

    create_sphere();
    Shader sphereShader("instancing.vs", "instancing.fs");
    unsigned int sphere_offset_buffer;
    glGenBuffers(1, &sphere_offset_buffer);
    glBindBuffer(GL_ARRAY_BUFFER, sphere_offset_buffer);
    glBufferData(GL_ARRAY_BUFFER, n_particles * sizeof(glm::vec3), &F_x_host[0], GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    unsigned int sphere_color_buffer;
    glGenBuffers(1, &sphere_color_buffer);
    glBindBuffer(GL_ARRAY_BUFFER, sphere_color_buffer);
    glBufferData(GL_ARRAY_BUFFER, n_particles * sizeof(glm::vec3), &F_colors_host[0], GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    unsigned int sphere_VAO, sphere_VBO;
    glGenVertexArrays(1, &sphere_VAO);
    glGenBuffers(1, &sphere_VBO);
    glBindVertexArray(sphere_VAO);
    glBindBuffer(GL_ARRAY_BUFFER, sphere_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(sphere_vertices), sphere_vertices, GL_STATIC_DRAW);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(1);
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)(3 * sizeof(float)));
    
    glEnableVertexAttribArray(2);
    glBindBuffer(GL_ARRAY_BUFFER, sphere_offset_buffer);
    glVertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
    glVertexAttribDivisor(2, 1);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glEnableVertexAttribArray(3);
    glBindBuffer(GL_ARRAY_BUFFER, sphere_color_buffer);
    glVertexAttribPointer(3, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
    glVertexAttribDivisor(3, 1);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    unsigned int sphere_EBO;
    glGenBuffers(1, &sphere_EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, sphere_EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(sphere_indices), sphere_indices, GL_STATIC_DRAW);

    sphereShader.use();
    glm::mat4 projection = glm::perspective(glm::radians(camera.Zoom), (float)SCR_WIDTH / (float)SCR_HEIGHT, 0.1f, 100.0f);
    sphereShader.setMat4("projection", projection);
    sphereShader.setVec3("lightPos", lightPos);
    sphereShader.setVec3("lightColor", lightColor);
    glm::mat4 model = glm::mat4(1.0f);
    sphereShader.setMat4("model", model);

    create_tool();
    Shader toolShader("tool.vs", "tool.fs");
    unsigned int tool_VAO, tool_VBO;
    glGenVertexArrays(1, &tool_VAO);
    glGenBuffers(1, &tool_VBO);
    glBindVertexArray(tool_VAO);
    glBindBuffer(GL_ARRAY_BUFFER, tool_VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(tool_vertices_host), tool_vertices_host, GL_STATIC_DRAW);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(1);
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)(3 * sizeof(float)));
    unsigned int tool_EBO;
    glGenBuffers(1, &tool_EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, tool_EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(tool_indices_host), tool_indices_host, GL_STATIC_DRAW);
    toolShader.use();
    toolShader.setMat4("projection", projection);
    toolShader.setVec3("lightPos", lightPos);
    toolShader.setVec3("lightColor", lightColor);
    toolShader.setVec3("objectColor", 0.7f, 0.8f, 0.7f);
    toolShader.setMat4("model", model);


    Shader mcShader("MCShader.vs", "MCShader.fs");
    unsigned int mc_VAO, mc_VBO, mc_EBO;
    glGenVertexArrays(1, &mc_VAO);
    glGenBuffers(1, &mc_VBO);
    glBindVertexArray(mc_VAO);
    glBindBuffer(GL_ARRAY_BUFFER, mc_VBO);
    glBufferData(GL_ARRAY_BUFFER,sizeof(MC_vertices_host), MC_vertices_host, GL_DYNAMIC_DRAW);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE,6 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(1);
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(float), (void*)(3 * sizeof(float)));
    glGenBuffers(1, &mc_EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, mc_EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(unsigned int) * n_MC_total_grid * 12, MC_indices, GL_DYNAMIC_DRAW);
    
    mcShader.setMat4("projection", projection);
    mcShader.setVec3("lightPos", lightPos);
    mcShader.setVec3("lightColor", lightColor);
    mcShader.setVec3("objectColor", 0.7f, 0.8f, 0.7f);
    mcShader.setMat4("model", model);

    int frame = 0;
    double render_time = 0.0;
    double grid_clear_time = 0.0;
    double p2g_time = 0.0;
    double boundary_condition_time = 0.0;
    double g2p_time = 0.0;

    checkCudaErrors(hipMemcpy(tool_center_device, tool_center_host, dim * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(tool_vertices_device, tool_vertices_host, 6 * tool_row * tool_col * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(tool_indices_device, tool_indices_host, 6 * (tool_row - 1) * tool_col * sizeof(unsigned int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(MC_et_device, MC_et_host, 256 * 12 * sizeof(int), hipMemcpyHostToDevice));
    init_particle << <n_particle_blocks, n_threads >> > (
        n_particles,
        F_x_device,
        F_v_device,
        F_C_device,
        F_dg_device,
        F_Jp_device,
        F_colors_device,
        F_materials_device,
        F_used_device
        );
    init_chuiti();
    checkCudaErrors(hipMemcpy(F_x_device, F_x_host, dim* n_particles * sizeof(float), hipMemcpyHostToDevice));

    init_rigid_particles << <n_rigid_blocks, n_threads >> > (
        n_rigid_particles,
        tool_vertices_device,
        tool_indices_device,
        tool_center_device,
        rigid_x_device
        );

    n_MC_triangle_host = new(int);
    while (!glfwWindowShouldClose(window)) {
        frame++;
        float currentFrame = static_cast<float>(glfwGetTime());
        deltaTime = currentFrame - lastFrame;
        lastFrame = currentFrame;
        processInput(window);
        glClearColor(0.1f, 0.1f, 0.1f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        
        for (int s = 0; s < step; s++) {
            if (tool_center_host[1] > -0.1) {
                tool_center_host[1] -= dt;
                checkCudaErrors(hipMemcpy(tool_center_device, tool_center_host, dim * sizeof(float), hipMemcpyHostToDevice));
                rigid_move << <n_rigid_blocks, n_threads >> > (
                    n_rigid_particles,
                    rigid_x_device,
                    dt
                    );
                tool_vertices_move << <n_tool_vertices, n_threads >> > (
                    n_tool_vertices,
                    tool_vertices_device,
                    dt
                    );
            }
            
            grid_clear << <n_grid_blocks, n_threads >> > (
                n_total_grid,
                F_grid_v_device,
                F_grid_m_device,
                C_grid_d_device,
                C_grid_A_device,
                C_grid_T_device,
                C_grid_surface_device
                );

            grid_CDF << <n_rigid_blocks, n_threads >> > (
                n_rigid_particles,
                tool_vertices_device,
                tool_indices_device,
                tool_center_device,
                rigid_x_device,
                dx,
                inv_dx,
                C_grid_d_device,
                C_grid_A_device,
                C_grid_T_device,
                C_grid_surface_device,
                n_grid
                );

            p2g << <n_particle_blocks, n_threads >> > (
                F_x_device,
                F_v_device,
                F_C_device,
                F_dg_device,
                F_Jp_device,
                F_colors_device,
                F_materials_device,
                F_grid_v_device,
                F_grid_m_device,
                F_used_device,
                dt,
                n_particles,
                dx,
                p_rho,
                p_vol,
                p_mass,
                nu,
                mu_0,
                lambda_0,
                WATER,
                JELLY,
                SNOW,
                C_grid_d_device,
                C_grid_A_device,
                C_grid_T_device,
                C_grid_surface_device,
                C_p_d_device,
                C_p_A_device,
                C_p_T_device,
                C_p_n_device,
                tool_center_device,
                n_grid
                );

            boundary_condition << <n_grid_blocks, n_threads >> > (
                n_total_grid,
                F_grid_v_device,
                F_grid_m_device,
                dt,
                n_grid,
                bound,
                inv_dx
                );

            g2p << <n_particle_blocks, n_threads >> > (
                F_x_device,
                F_C_device,
                F_v_device,
                F_colors_device,
                F_materials_device,
                F_used_device,
                F_grid_v_device,
                F_grid_m_device,
                n_particles,
                n_grid,
                dx,
                dt,
                C_grid_d_device,
                C_grid_A_device,
                C_grid_T_device,
                C_grid_surface_device,
                C_p_d_device,
                C_p_A_device,
                C_p_T_device,
                C_p_n_device,
                tool_center_device,
                p_mass    
                );
        }
        
        checkCudaErrors(hipMemcpy(F_x_host, F_x_device, dim * n_particles * sizeof(float), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(F_colors_host, F_colors_device, dim * n_particles * sizeof(float), hipMemcpyDeviceToHost));
        //checkCudaErrors(hipMemcpy(tool_center_host, tool_center_device, dim * sizeof(float), hipMemcpyDeviceToHost));
        glm::mat4 view = camera.GetViewMatrix();
        sphereShader.use();
        sphereShader.setMat4("view", view);
        glBindVertexArray(sphere_VAO);
        glBindBuffer(GL_ARRAY_BUFFER, sphere_offset_buffer);
        glBufferSubData(GL_ARRAY_BUFFER, 0, n_particles * sizeof(glm::vec3), F_x_host);
        glBindBuffer(GL_ARRAY_BUFFER, sphere_color_buffer);
        glBufferSubData(GL_ARRAY_BUFFER, 0, n_particles * sizeof(glm::vec3), F_colors_host);
        //glDrawElementsInstanced(GL_TRIANGLES, 6 * (sphere_row - 1) * sphere_col, GL_UNSIGNED_INT, 0, n_particles);

        toolShader.use();
        toolShader.setMat4("view", view);
        toolShader.setVec3("aOffset", tool_center_host[0] - tool_center_init[0], tool_center_host[1] - tool_center_init[1], tool_center_host[2] - tool_center_init[2]);
        glBindVertexArray(tool_VAO);
       glDrawElements(GL_TRIANGLES, 6 * (tool_row - 1) * tool_col, GL_UNSIGNED_INT, 0);

        checkCudaErrors(hipMemcpy(rigid_x_host, rigid_x_device, dim * n_rigid_particles * sizeof(float), hipMemcpyDeviceToHost));
        sphereShader.use();
        sphereShader.setMat4("view", view);
        glBindVertexArray(sphere_VAO);
        glBindBuffer(GL_ARRAY_BUFFER, sphere_offset_buffer);
        glBufferSubData(GL_ARRAY_BUFFER, 0, n_rigid_particles * sizeof(glm::vec3), rigid_x_host);
        glBindBuffer(GL_ARRAY_BUFFER, sphere_color_buffer);
        glBufferSubData(GL_ARRAY_BUFFER, 0, n_particles * sizeof(glm::vec3), F_colors_host);
        glDrawElementsInstanced(GL_TRIANGLES, 6 * (sphere_row - 1) * sphere_col, GL_UNSIGNED_INT, 0, n_rigid_particles);


        
        *n_MC_triangle_host = 0;
        //std::cout << *n_MC_triangle_host << std::endl;
        checkCudaErrors(hipMemcpy(n_MC_triangle_device, n_MC_triangle_host, sizeof(int), hipMemcpyHostToDevice));
        compute_implicit_face << <n_MC_grid_blocks, n_threads >> > (
            n_MC_grid,
            n_MC_total_grid,
            n_particles,
            MC_dx,
            F_x_device,
            MC_SDF_device
            );
        implicit_to_explicit << <n_MC_grid_blocks, n_threads >> > (
            n_MC_grid,
            n_MC_total_grid,
            MC_SDF_device,
            n_MC_triangle_device,
            MC_et_device,
            MC_vertices_device,
            MC_dx
            );
        checkCudaErrors(hipMemcpy(MC_vertices_host, MC_vertices_device, dim * 15 * n_MC_total_grid *sizeof(float), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(n_MC_triangle_host, n_MC_triangle_device, sizeof(int), hipMemcpyDeviceToHost));
       /* sphereShader.use();
        sphereShader.setMat4("view", view);
        sphereShader.use();
        sphereShader.setMat4("view", view);
        glBindVertexArray(sphere_VAO);
        glBindBuffer(GL_ARRAY_BUFFER, sphere_offset_buffer);
        glBufferSubData(GL_ARRAY_BUFFER, 0, n_particles * sizeof(glm::vec3), MC_vertices_host);
        glBindBuffer(GL_ARRAY_BUFFER, sphere_color_buffer);
        glBufferSubData(GL_ARRAY_BUFFER, 0, n_particles * sizeof(glm::vec3), F_colors_host);
        glDrawElementsInstanced(GL_TRIANGLES, 6 * (sphere_row - 1) * sphere_col, GL_UNSIGNED_INT, 0, n_particles);*/

        mcShader.use();
        projection = glm::perspective(glm::radians(camera.Zoom), (float)SCR_WIDTH / (float)SCR_HEIGHT, 0.1f, 100.0f);
        mcShader.setMat4("projection", projection);
        mcShader.setVec3("lightPos", lightPos);
        mcShader.setVec3("lightColor", lightColor);
        mcShader.setVec3("objectColor", 0.8f, 0.3f, 0.3f);
        model = glm::mat4(1.0f);
        mcShader.setMat4("model", model);
        view = camera.GetViewMatrix();
        mcShader.setMat4("view", view);
        
        smooth_normal();
        glBindVertexArray(mc_VAO);
        glBindBuffer(GL_ARRAY_BUFFER, mc_VBO);
        glBufferSubData(GL_ARRAY_BUFFER, 0, 3 * *n_MC_triangle_host * 6 *sizeof(float), MC_vertices_host);
        ////glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
        //glDrawArrays(GL_TRIANGLES, 0, 3 * *n_MC_triangle_host);
        glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, mc_EBO);
        glBufferSubData(GL_ELEMENT_ARRAY_BUFFER, 0, sizeof(unsigned int)* 3 * *n_MC_triangle_host, MC_indices);
        glDrawElements(GL_TRIANGLES, 3 * *n_MC_triangle_host, GL_UNSIGNED_INT, 0);
            
        
        glfwSwapBuffers(window);
        glfwPollEvents();
        
    }
    printf("------------------------------------\n");
    printf("-grid_clear:                %.6f\n", grid_clear_time / frame);
    printf("-p2g:                       %.6f\n", p2g_time / frame);
    printf("-boundary_condition:        %.6f\n", boundary_condition_time / frame);
    printf("-g2p:                       %.6f\n", g2p_time / frame);
    printf("-render:                    %.6f\n", render_time / frame);
    printf("-total:                    %.6f\n", (grid_clear_time + p2g_time + boundary_condition_time + g2p_time + render_time) / frame);
    printf("------------------------------------\n");

    checkCudaErrors(hipFree(F_x_device));
    checkCudaErrors(hipFree(F_v_device));
    checkCudaErrors(hipFree(F_C_device));
    checkCudaErrors(hipFree(F_dg_device));
    checkCudaErrors(hipFree(F_Jp_device));
    checkCudaErrors(hipFree(F_colors_device));
    checkCudaErrors(hipFree(F_materials_device));
    checkCudaErrors(hipFree(F_grid_v_device));
    checkCudaErrors(hipFree(F_grid_m_device));
    checkCudaErrors(hipFree(F_used_device));

    checkCudaErrors(hipFree(C_grid_d_device));
    checkCudaErrors(hipFree(C_grid_A_device));
    checkCudaErrors(hipFree(C_grid_T_device));
    checkCudaErrors(hipFree(C_grid_surface_device));
    checkCudaErrors(hipFree(C_p_d_device));
    checkCudaErrors(hipFree(C_p_A_device));
    checkCudaErrors(hipFree(C_p_T_device));
    checkCudaErrors(hipFree(C_p_n_device));

    checkCudaErrors(hipFree(tool_vertices_device));
    checkCudaErrors(hipFree(tool_indices_device));
    checkCudaErrors(hipFree(tool_center_device));
    checkCudaErrors(hipFree(rigid_x_device));

    checkCudaErrors(hipFree(MC_SDF_device));
    checkCudaErrors(hipFree(n_MC_triangle_device));
    checkCudaErrors(hipFree(MC_et_device));
    checkCudaErrors(hipFree(MC_vertices_device));

    free(n_MC_triangle_host);
 
    glDeleteVertexArrays(1, &sphere_VAO);
    glDeleteBuffers(1, &sphere_VBO);
    glDeleteBuffers(1, &sphere_EBO);
    glDeleteBuffers(1, &sphere_offset_buffer);
    glDeleteBuffers(1, &sphere_color_buffer);
    glDeleteVertexArrays(1, &tool_VAO);
    glDeleteBuffers(1, &tool_VBO);
    glDeleteBuffers(1, &tool_EBO);
    glfwTerminate();

    return 0;
}


void create_sphere() {
    float r = 0.008;
    for (int i = 0; i < sphere_row; i++) {
        float theta = i * glm::pi<float>() / sphere_row;
        for (int j = 0; j < sphere_col; j++) {
            float phi = j * 2 * glm::pi<float>() / sphere_col;
            float x = r * sinf(theta) * cosf(phi);
            float y = r * sinf(theta) * sinf(phi);
            float z = r * cosf(theta);
            int index = (i * sphere_col + j) * 6;
            sphere_vertices[index + 0] = x;
            sphere_vertices[index + 1] = y;
            sphere_vertices[index + 2] = z;
            glm::vec3 norm = glm::normalize(glm::vec3(x, y, z));
            sphere_vertices[index + 3] = norm.x;
            sphere_vertices[index + 4] = norm.y;
            sphere_vertices[index + 5] = norm.z;
        }
    }
    for (int i = 0; i < sphere_row - 1; i++) {
        for (int j = 0; j < sphere_col; j++) {
            int index = (i * sphere_col + j) * 6;
            int temp = i * sphere_col + j;
            sphere_indices[index + 0] = temp;
            if ((temp + 1) % sphere_col == 0) {
                sphere_indices[index + 1] = i * sphere_col;
                sphere_indices[index + 2] = (i + 1) * sphere_col;
            }
            else {
                sphere_indices[index + 1] = temp + 1;
                sphere_indices[index + 2] = temp + 1 + sphere_col;
            }
            sphere_indices[index + 3] = temp;
            sphere_indices[index + 4] = sphere_indices[index + 2];
            sphere_indices[index + 5] = temp + sphere_col;
        }
    }
}

void create_tool() {
    float r = 0.1;
    //float center[3]{ 0.0f, 0.0f, 0.0f };
    for (int i = 0; i < tool_row; i++) {
        float phi = (8.0 - i) / 14.0 * glm::pi<float>();
        for (int j = 0; j < tool_col; j++) {
            int index = (i * tool_col + j) * 6;
            float theta = 2.0f * j / tool_col * glm::pi<float>();
            float x = r * sin(phi) * cos(theta);
            float z = r * sin(phi) * sin(theta);
            float y = r * cos(phi);
            tool_vertices_host[index + 0] = x + tool_center_host[0];
            tool_vertices_host[index + 1] = y + tool_center_host[1];
            tool_vertices_host[index + 2] = z + tool_center_host[2];
            glm::vec3 norm = glm::normalize(glm::vec3(x, y, z));
            tool_vertices_host[index + 3] = norm.x;
            tool_vertices_host[index + 4] = norm.y;
            tool_vertices_host[index + 5] = norm.z;
        }
    }
    for (int i = 0; i < tool_row - 1; i++) {
        for (int j = 0; j < tool_col; j++) {
            int index = (i * tool_col + j) * 6;
            int temp = i * tool_col + j;
            tool_indices_host[index + 0] = temp;
            if ((temp + 1) % tool_col == 0) {
                tool_indices_host[index + 1] = i * tool_col;
                tool_indices_host[index + 2] = (i + 1) * tool_col;
            }
            else {
                tool_indices_host[index + 1] = temp + 1;
                tool_indices_host[index + 2] = temp + 1 + tool_col;
            }
            tool_indices_host[index + 3] = temp;
            tool_indices_host[index + 4] = tool_indices_host[index + 2];
            tool_indices_host[index + 5] = temp + tool_col;
        }
    }
}

